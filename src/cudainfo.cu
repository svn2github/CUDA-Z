/*!
	\file cudainfo.cu
	\brief CUDA information functions.
	\author AG
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <windows.h>

#include <QDebug>

#include "cudainfo.h"

#define CZ_BAND_BUF_SIZE	(32 * (1 << 20))	/*!< Transfer buffer size. */
#define CZ_BAND_LOOPS_NUM	(10)			/*!< Number of loops to run transfer test to. */

/*
	\brief Prototype of function \a hipDeviceGetAttribute().
*/
typedef hipError_t (CUDAAPI *cuDeviceGetAttribute_t)(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);

/*
	\brief Pointer to function \a hipDeviceGetAttribute().
	This parameter is initializaed by #cudaIsInit().
*/
static cuDeviceGetAttribute_t p_cuDeviceGetAttribute = NULL;

/*
	\brief Check if CUDa fully initialized.
	This function loads nvcuda.dll and finds function \a hipDeviceGetAttribute.
	\return \a true in case of success, \a false in case of error.
*/
static bool cudaIsInit(void) {

	HINSTANCE hDll;

//	printf("cudaIsInit called\n");

	if(p_cuDeviceGetAttribute == NULL) {

//		printf("load nvcuda.dll\n");

		hDll = LoadLibrary(L"nvcuda.dll");
		if(hDll == NULL) {
			return false;
		}

//		printf("getting hipDeviceGetAttribute\n");

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)GetProcAddress(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
//			printf("flail hipDeviceGetAttribute\n");
			return false;
		}

//		printf("got hipDeviceGetAttribute at 0x%08X\n", p_cuDeviceGetAttribute);
	}

	return true;
}

/*!
	\brief Check how many CUDA-devices are present.
	\return number of CUDA-devices in case of success, \a 0 if no CUDA-devies were found.
*/
int cudaDeviceFound(void) {

	int count;

	if(hipGetDeviceCount(&count) == hipSuccess) {
		return count;
	}
	return 0;
}

/*!
	\brief Read information about a CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaReadDeviceInfo(
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int num				/*!< Number (index) of CUDA-device. */
) {
	hipDeviceProp_t prop;
	int overlap;


	if(info == NULL)
		return -1;

	if(!cudaIsInit())
		return -1;

	if(num >= cudaDeviceFound())
		return -1;

	if(hipGetDeviceProperties(&prop, num) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return -1;
	}

	if(p_cuDeviceGetAttribute(&overlap, hipDeviceAttributeAsyncEngineCount, num) != hipSuccess)
		return -1;

	info->num = num;
	strcpy(info->deviceName, prop.name);
	info->major = prop.major;
	info->minor = prop.minor;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.SIMDWidth = prop.warpSize;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate;

	info->mem.totalGlobal = prop.totalGlobalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstMem;
	info->mem.textureAlignment = prop.textureAlignment;
	info->mem.gpuOverlap = overlap;

	return 0;
}

/*!
	\brief Global service data structure for bandwith calulations.
*/
struct CZDeviceInfoBandGlobalData {
	void		*memHostPage;	/*!< Pageable host memory. */
	void		*memHostPin;	/*!< Pinned host memory. */
};

/*!
	\brief Local service data structure for bandwith calulations.
*/
struct CZDeviceInfoBandLocalData {
	void		*memDevice1;	/*!< Device memory. */
	void		*memDevice2;	/*!< Device memory. */
};

/*!
	\brief Global service data.
*/
CZDeviceInfoBandGlobalData *globalData = NULL;

/*!
	\brief Allocate buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthAlloc(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandGlobalData *gData;
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	if(globalData == NULL) {

		printf("Alloc global buffers.\n");

		gData = (CZDeviceInfoBandGlobalData*)malloc(sizeof(gData));
		if(gData == NULL) {
			return -1;
		}

		printf("Alloc host pageable.\n");

		gData->memHostPage = (void*)malloc(CZ_BAND_BUF_SIZE);
		if(gData->memHostPage == NULL) {
			free(gData);
			return -1;
		}

		printf("Host pageable is at 0x%08X.\n", gData->memHostPage);

		printf("Alloc host pinned.\n");

		if(hipHostMalloc((void**)&gData->memHostPin, CZ_BAND_BUF_SIZE) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			free(gData->memHostPage);
			free(gData);
			return -1;
		}

		printf("Host pinned is at 0x%08X.\n", gData->memHostPin);

		globalData = gData;
	}

	if(info->band.localData == NULL) {

		printf("Selecting %s.\n", info->deviceName);

		if(hipSetDevice(info->num) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			return -1;
		}

		printf("Alloc local buffers for %s.\n", info->deviceName);

		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(lData));
		if(lData == NULL) {
			return -1;
		}

		printf("Alloc local buffer 1 for %s.\n", info->deviceName);

		if(hipMalloc((void**)&lData->memDevice1, CZ_BAND_BUF_SIZE) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			free(lData);
			return -1;
		}

		printf("Host buffer 1 is at 0x%08X.\n", lData->memDevice1);

		printf("Alloc local buffer 2 for %s.\n", info->deviceName);

		if(hipMalloc((void**)&lData->memDevice2, CZ_BAND_BUF_SIZE) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			hipFree(lData->memDevice1);
			free(lData);
			return -1;
		}

		printf("Host buffer 2 is at 0x%08X.\n", lData->memDevice2);

		info->band.localData = (void*)lData;
	}

	return 0;
}

/*!
	\brief Free buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthFree(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandGlobalData *gData;
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	gData = globalData;
	if(gData != NULL) {

		printf("Free host pageable.\n");

		if(gData->memHostPage != NULL)
			free(gData->memHostPage);

		printf("Free host pinned.\n");

		if(gData->memHostPin != NULL)
			hipHostFree(gData->memHostPin);

		printf("Free global buffers.\n");

		free(gData);
	}
	globalData = NULL;

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	if(lData != NULL) {

		printf("Selecting %s.\n", info->deviceName);

		if(hipSetDevice(info->num) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			return -1;
		}

		printf("Free local buffer 1 for %s.\n", info->deviceName);

		if(lData->memDevice1 != NULL)
			hipFree(lData->memDevice1);

		printf("Free local buffer 2 for %s.\n", info->deviceName);

		if(lData->memDevice2 != NULL)
			hipFree(lData->memDevice2);


		printf("Free local buffers for %s.\n", info->deviceName);

		free(lData);
	}
	info->band.localData = NULL;

	return 0;
}

/*!
	\brief Reset results of bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthReset(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->band.copyHDPage = 0;
	info->band.copyHDPin = 0;
	info->band.copyDHPage = 0;
	info->band.copyDHPin = 0;
	info->band.copyDD = 0;

	return 0;
}

int cudaCalcDeviceBandwidthTestHD (
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int pinned
) {
	CZDeviceInfoBandGlobalData *gData;
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	if(hipSetDevice(info->num) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return 0;
	}

	if(hipEventCreate(&start) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return 0;
	}

	if(hipEventCreate(&stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		return 0;
	}

	gData = globalData;
	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? gData->memHostPin: gData->memHostPage;
	memDevice = lData->memDevice1;

	printf("Starting H(0x%08X)->D(0x%08X) test (%s) on %s.\n",
		memHost, memDevice,
		pinned? "pinned": "pageable",
		info->deviceName);

	if(hipEventRecord(start, 0) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}
	for(i = 0; i < CZ_BAND_LOOPS_NUM; i++) {
		if(hipMemcpy(memDevice, memHost, CZ_BAND_BUF_SIZE, hipMemcpyHostToDevice) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}
	}
	if(hipEventRecord(stop, 0) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	if(hipEventSynchronize(stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	if(hipEventElapsedTime(&timeMs, start, stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (1000 * (float)CZ_BAND_BUF_SIZE * (float)CZ_BAND_LOOPS_NUM) / (timeMs * (float)(1 << 10));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

int cudaCalcDeviceBandwidthTestDH (
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int pinned
) {
	CZDeviceInfoBandGlobalData *gData;
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	if(hipSetDevice(info->num) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return 0;
	}

	if(hipEventCreate(&start) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return 0;
	}

	if(hipEventCreate(&stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		return 0;
	}

	gData = globalData;
	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? gData->memHostPin: gData->memHostPage;
	memDevice = lData->memDevice2;

	printf("Starting D(0x%08X)->H(0x%08X) test (%s) on %s.\n",
		memDevice, memHost,
		pinned? "pinned": "pageable",
		info->deviceName);

	if(hipEventRecord(start, 0) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}
	for(i = 0; i < CZ_BAND_LOOPS_NUM; i++) {
		if(hipMemcpy(memDevice, memHost, CZ_BAND_BUF_SIZE, hipMemcpyHostToDevice) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}
	}
	if(hipEventRecord(stop, 0) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	if(hipEventSynchronize(stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	if(hipEventElapsedTime(&timeMs, start, stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (1000 * (float)CZ_BAND_BUF_SIZE * (float)CZ_BAND_LOOPS_NUM) / (timeMs * (float)(1 << 10));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

int cudaCalcDeviceBandwidthTestDD (
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
//	CZDeviceInfoBandGlobalData *gData;
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memDevice1;
	void *memDevice2;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	if(hipSetDevice(info->num) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return 0;
	}

	if(hipEventCreate(&start) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		return 0;
	}

	if(hipEventCreate(&stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		return 0;
	}

//	gData = globalData;
	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memDevice1 = lData->memDevice1;
	memDevice2 = lData->memDevice2;

	printf("Starting D(0x%08X)->D(0x%08X) test on %s.\n",
		memDevice1, memDevice2,
		info->deviceName);

	if(hipEventRecord(start, 0) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}
	for(i = 0; i < CZ_BAND_LOOPS_NUM; i++) {
		if(hipMemcpy(memDevice2, memDevice1, CZ_BAND_BUF_SIZE, hipMemcpyDeviceToDevice) != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}
	}
	if(hipEventRecord(stop, 0) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	if(hipEventSynchronize(stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	if(hipEventElapsedTime(&timeMs, start, stop) != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0;
	}

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (1000 * (float)CZ_BAND_BUF_SIZE * (float)CZ_BAND_LOOPS_NUM) / (timeMs * (float)(1 << 10));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run several bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthTest(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	info->band.copyHDPage = cudaCalcDeviceBandwidthTestHD(info, 0);
	info->band.copyHDPin = cudaCalcDeviceBandwidthTestHD(info, 1);
	info->band.copyDHPage = cudaCalcDeviceBandwidthTestDH(info, 0);
	info->band.copyDHPin = cudaCalcDeviceBandwidthTestDH(info, 1);
	info->band.copyDD = cudaCalcDeviceBandwidthTestDD(info);

	return 0;
}

/*!
	\brief Calculate bandwidth information about CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidth(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

//	printf("cudaCalcDeviceBandwidth called!\n");

	if(info == NULL)
		return -1;

	if(cudaCalcDeviceBandwidthReset(info) != 0)
		return -1;

	if(!cudaIsInit())
		return -1;

	if(cudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	if(cudaCalcDeviceBandwidthTest(info) != 0)
		return -1;

	return 0;
}

/*!
	\brief Cleanup after test and bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCleanDevice(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(cudaCalcDeviceBandwidthFree(info) != 0)
		return -1;

	return 0;
}
