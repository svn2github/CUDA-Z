#include "hip/hip_runtime.h"
/*!
	\file cudainfo.cu
	\brief CUDA information functions.
	\author AG
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>

//#include <QDebug>
#include <qglobal.h>
#define printf(fmt, ...)

#include "cudainfo.h"

#define CZ_COPY_BUF_SIZE	(16 * (1 << 20))	/*!< Transfer buffer size. */
#define CZ_COPY_LOOPS_NUM	8			/*!< Number of loops to run transfer test to. */

#define CZ_CALC_LOOPS_NUM	8			/*!< Number of loops to run calculation loop. */
#define CZ_CALC_THREADS_NUM	65536			/*!< Number of threads to run calculation loop. */
#define CZ_CALC_BLOCK_SIZE	128			/*!< Size of instruction block. */
#define CZ_CALC_BLOCK_NUM	16			/*!< Number of instruction blocks in loop. */
#define CZ_CALC_OPS_NUM		2			/*!< Number of operations per one loop. */

/*!
	\brief Error handling of CUDA RT calls.
*/
#define CZ_CUDA_CALL(funcCall, errProc) \
	{ \
		hipError_t errCode; \
		if((errCode = (funcCall)) != hipSuccess) { \
			printf("CUDA Error: %s\n", hipGetErrorString(errCode)); \
			errProc; \
		} \
	}

/*!
	\brief Prototype of function \a hipDeviceGetAttribute().
*/
typedef hipError_t (CUDAAPI *cuDeviceGetAttribute_t)(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);

/*!
	\brief Prototype of function \a hipInit().
*/
typedef hipError_t (CUDAAPI *cuInit_t)(unsigned int Flags);

/*!
	\brief Pointer to function \a hipDeviceGetAttribute().
	This parameter is initializaed by CZCudaIsInit().
*/
static cuDeviceGetAttribute_t p_cuDeviceGetAttribute = NULL;

/*!
	\brief Pointer to function \a hipInit().
	This parameter is initializaed by CZCudaIsInit().
*/
static cuInit_t p_cuInit = NULL;

#ifdef Q_OS_WIN
#include <windows.h>
/*!
	\brief Check if CUDA fully initialized.
	This function loads nvcuda.dll and finds functions \a hipInit()
	and \a hipDeviceGetAttribute().
	\return \a true in case of success, \a false in case of error.
*/
static bool CZCudaIsInit(void) {

	HINSTANCE hDll;

	if((p_cuInit == NULL) || (p_cuDeviceGetAttribute == NULL)) {

		hDll = LoadLibraryA("nvcuda.dll");
		if(hDll == NULL) {
			return false;
		}

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)GetProcAddress(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
			return false;
		}

		p_cuInit = (cuInit_t)GetProcAddress(hDll, "hipInit");
		if(p_cuInit == NULL) {
			return false;
		}
	}

	return true;
}
#else//!Q_OS_WIN
#error Function CZCudaIsInit() is not implemented for your platform!
#endif//Q_OS_WIN


/*!
	\brief Check if CUDA is present here.
*/
bool CZCudaCheck(void) {

	if(!CZCudaIsInit())
		return false;

	if(p_cuInit(0) == hipErrorNotInitialized) {
		return false;
	}

	return true;
}

/*!
	\brief Check how many CUDA-devices are present.
	\return number of CUDA-devices in case of success, \a 0 if no CUDA-devies were found.
*/
int CZCudaDeviceFound(void) {

	int count;

	CZ_CUDA_CALL(hipGetDeviceCount(&count),
		return 0);

	return count;
}

/*!
	\brief Read information about a CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaReadDeviceInfo(
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int num				/*!< Number (index) of CUDA-device. */
) {
	hipDeviceProp_t prop;
	int overlap;

	if(info == NULL)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(num >= CZCudaDeviceFound())
		return -1;

	CZ_CUDA_CALL(hipGetDeviceProperties(&prop, num),
		return -1);

	if(p_cuDeviceGetAttribute(&overlap, hipDeviceAttributeAsyncEngineCount, num) != hipSuccess)
		return -1;

	info->num = num;
	strcpy(info->deviceName, prop.name);
	info->major = prop.major;
	info->minor = prop.minor;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.SIMDWidth = prop.warpSize;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate;

	info->mem.totalGlobal = prop.totalGlobalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstMem;
	info->mem.textureAlignment = prop.textureAlignment;
	info->mem.gpuOverlap = overlap;

	return 0;
}

/*!
	\brief Local service data structure for bandwith calulations.
*/
struct CZDeviceInfoBandLocalData {
	void		*memHostPage;	/*!< Pageable host memory. */
	void		*memHostPin;	/*!< Pinned host memory. */
	void		*memDevice1;	/*!< Device memory buffer 1. */
	void		*memDevice2;	/*!< Device memory buffer 2. */
};

/*!
	\brief Allocate buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthAlloc(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	if(info->band.localData == NULL) {

		printf("Selecting %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipSetDevice(info->num),
			return -1);

		printf("Alloc local buffers for %s.\n", info->deviceName);

		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(*lData));
		if(lData == NULL) {
			return -1;
		}

		printf("Alloc host pageable for %s.\n", info->deviceName);

		lData->memHostPage = (void*)malloc(CZ_COPY_BUF_SIZE);
		if(lData->memHostPage == NULL) {
			free(lData);
			return -1;
		}

		printf("Host pageable is at 0x%08X.\n", lData->memHostPage);

		printf("Alloc host pinned for %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipHostMalloc((void**)&lData->memHostPin, CZ_COPY_BUF_SIZE),
			free(lData->memHostPage);
			free(lData);
			return -1);

		printf("Host pinned is at 0x%08X.\n", lData->memHostPin);

		printf("Alloc device buffer 1 for %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice1, CZ_COPY_BUF_SIZE),
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		printf("Device buffer 1 is at 0x%08X.\n", lData->memDevice1);

		printf("Alloc device buffer 2 for %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice2, CZ_COPY_BUF_SIZE),
			hipFree(lData->memDevice1);
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		printf("Device buffer 2 is at 0x%08X.\n", lData->memDevice2);

		info->band.localData = (void*)lData;
	}

	return 0;
}

/*!
	\brief Free buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthFree(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	if(lData != NULL) {

		printf("Selecting %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipSetDevice(info->num),
			return -1);

		printf("Free host pageable for %s.\n", info->deviceName);

		if(lData->memHostPage != NULL)
			free(lData->memHostPage);

		printf("Free host pinned for %s.\n", info->deviceName);

		if(lData->memHostPin != NULL)
			hipHostFree(lData->memHostPin);

		printf("Free device buffer 1 for %s.\n", info->deviceName);

		if(lData->memDevice1 != NULL)
			hipFree(lData->memDevice1);

		printf("Free device buffer 2 for %s.\n", info->deviceName);

		if(lData->memDevice2 != NULL)
			hipFree(lData->memDevice2);

		printf("Free local buffers for %s.\n", info->deviceName);

		free(lData);
	}
	info->band.localData = NULL;

	return 0;
}

/*!
	\brief Reset results of bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthReset(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->band.copyHDPage = 0;
	info->band.copyHDPin = 0;
	info->band.copyDHPage = 0;
	info->band.copyDHPin = 0;
	info->band.copyDD = 0;

	return 0;
}

/*!
	\brief Run host to device data transfer bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static float CZCudaCalcDeviceBandwidthTestHD (
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int pinned			/*!< Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice = lData->memDevice1;

	printf("Starting H(0x%08X)->D(0x%08X) test (%s) on %s.\n",
		memHost, memDevice,
		pinned? "pinned": "pageable",
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	for(i = 0; i < CZ_COPY_LOOPS_NUM; i++) {
		CZ_CUDA_CALL(hipMemcpy(memDevice, memHost, CZ_COPY_BUF_SIZE, hipMemcpyHostToDevice),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (
		1000 *
		(float)CZ_COPY_BUF_SIZE *
		(float)CZ_COPY_LOOPS_NUM
	) / (
		timeMs *
		(float)(1 << 10)
	);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run device to host data transfer bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static float CZCudaCalcDeviceBandwidthTestDH (
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int pinned			/*!< Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice = lData->memDevice2;

	printf("Starting D(0x%08X)->H(0x%08X) test (%s) on %s.\n",
		memDevice, memHost,
		pinned? "pinned": "pageable",
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	for(i = 0; i < CZ_COPY_LOOPS_NUM; i++) {
		CZ_CUDA_CALL(hipMemcpy(memDevice, memHost, CZ_COPY_BUF_SIZE, hipMemcpyHostToDevice),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (
		1000 *
		(float)CZ_COPY_BUF_SIZE *
		(float)CZ_COPY_LOOPS_NUM
	) / (
		timeMs *
		(float)(1 << 10)
	);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run device to device data transfer bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static float CZCudaCalcDeviceBandwidthTestDD (
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memDevice1;
	void *memDevice2;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memDevice1 = lData->memDevice1;
	memDevice2 = lData->memDevice2;

	printf("Starting D(0x%08X)->D(0x%08X) test on %s.\n",
		memDevice1, memDevice2,
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	for(i = 0; i < CZ_COPY_LOOPS_NUM; i++) {
		CZ_CUDA_CALL(hipMemcpy(memDevice2, memDevice1, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToDevice),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (
		1000 *
		(float)CZ_COPY_BUF_SIZE *
		(float)CZ_COPY_LOOPS_NUM
	) / (
		timeMs *
		(float)(1 << 10)
	);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run several bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthTest(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	info->band.copyHDPage = CZCudaCalcDeviceBandwidthTestHD(info, 0);
	info->band.copyHDPin = CZCudaCalcDeviceBandwidthTestHD(info, 1);
	info->band.copyDHPage = CZCudaCalcDeviceBandwidthTestDH(info, 0);
	info->band.copyDHPin = CZCudaCalcDeviceBandwidthTestDH(info, 1);
	info->band.copyDD = CZCudaCalcDeviceBandwidthTestDD(info);

	return 0;
}

/*!
	\brief Prepare buffers bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaPrepareDevice(
	struct CZDeviceInfo *info
) {

	if(info == NULL)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	return 0;
}

/*!
	\brief Calculate bandwidth information about CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaCalcDeviceBandwidth(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

//	printf("CZCudaCalcDeviceBandwidth called!\n");

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthReset(info) != 0)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	if(CZCudaCalcDeviceBandwidthTest(info) != 0)
		return -1;

	return 0;
}

/*!
	\brief Cleanup after test and bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaCleanDevice(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthFree(info) != 0)
		return -1;

	return 0;
}

/*!
	\brief Reset results of preformance calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDevicePerformanceReset(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->perf.calcFloat = 0;
	info->perf.calcInteger32 = 0;
	info->perf.calcInteger24 = 0;

	return 0;
}

/*!
	\brief 16 MAD instructions for float point test.
*/
#define CZ_CALC_FMAD_16(a, b) \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \

/*!
	\brief 128 MAD instructions for float point test.
*/
#define CZ_CALC_FMAD_128(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \

/*!
	\brief 16 MAD instructions for 32-bit integer test.
*/
#define CZ_CALC_IMAD32_16(a, b) \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \

/*!
	\brief 128 MAD instructions for 32-bit integer test.
*/
#define CZ_CALC_IMAD32_128(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \

/*!
	\brief 16 MAD instructions for 24-bit integer test.
*/
#define CZ_CALC_IMAD24_16(a, b) \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \

/*!
	\brief 128 MAD instructions for 24-bit integer test.
*/
#define CZ_CALC_IMAD24_128(a, b) \
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\

#define CZ_CALC_MODE_FLOAT	0	/*!< Float point test mode. */
#define CZ_CALC_MODE_INTEGER32	1	/*!< 32-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER24	2	/*!< 24-bit integer test mode. */

/*!
	\brief GPU code for float point test.
*/
static __global__ void CZCudaCalcKernelFloat(void *buf) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float *arr = (float*)buf;
	float val1 = index;
	float val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_LOOPS_NUM; i++) {
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
		CZ_CALC_FMAD_128(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!
	\brief GPU code for 32-bit integer test.
*/
static __global__ void CZCudaCalcKernelInteger32(void *buf) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_LOOPS_NUM; i++) {
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
		CZ_CALC_IMAD32_128(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!
	\brief GPU code for 24-bit integer test.
*/
static __global__ void CZCudaCalcKernelInteger24(void *buf) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_LOOPS_NUM; i++) {
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
		CZ_CALC_IMAD24_128(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!
	\brief Run GPU calculation performace tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static float CZCudaCalcDevicePerformanceTest(
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int mode			/*!< Run performance test in one of modes. */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float performanceKOPs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	printf("Starting %s test on %s.\n",
		(mode == CZ_CALC_MODE_FLOAT)? "float point":
		(mode == CZ_CALC_MODE_INTEGER32)? "32-bit integer":
		(mode == CZ_CALC_MODE_INTEGER24)? "24-bit integer": "unknown",
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	switch(mode) {
	case CZ_CALC_MODE_FLOAT:
		CZCudaCalcKernelFloat<<<CZ_CALC_THREADS_NUM / info->core.maxThreadsPerBlock, info->core.maxThreadsPerBlock>>>(lData->memDevice1);
		break;

	case CZ_CALC_MODE_INTEGER32:
		CZCudaCalcKernelInteger32<<<CZ_CALC_THREADS_NUM / info->core.maxThreadsPerBlock, info->core.maxThreadsPerBlock>>>(lData->memDevice1);
		break;

	case CZ_CALC_MODE_INTEGER24:
		CZCudaCalcKernelInteger24<<<CZ_CALC_THREADS_NUM / info->core.maxThreadsPerBlock, info->core.maxThreadsPerBlock>>>(lData->memDevice1);
		break;
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	performanceKOPs = (
		(float)CZ_CALC_THREADS_NUM * 
		(float)CZ_CALC_LOOPS_NUM * 
		(float)CZ_CALC_OPS_NUM *
		(float)CZ_CALC_BLOCK_SIZE *
		(float)CZ_CALC_BLOCK_NUM
	) / (float)timeMs;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)performanceKOPs;
}

/*!
	\brief Calculate performance information about CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaCalcDevicePerformance(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

//	printf("CZCudaCalcDevicePerformance called!\n");

	if(info == NULL)
		return -1;

	if(CZCudaCalcDevicePerformanceReset(info) != 0)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	info->perf.calcFloat = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_FLOAT);
	info->perf.calcInteger32 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER32);
	info->perf.calcInteger24 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER24);

	return 0;
}
