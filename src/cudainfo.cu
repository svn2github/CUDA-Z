/*!
	\file cudainfo.cu
	\brief CUDA information functions.
	\author AG
*/

#include <hip/hip_runtime.h>
#include <string.h>

#include "cudainfo.h"

/*!
	\brief Check if CUDA is present here.
*/
bool cudaCheck(void) {

	if(hipInit(0) == hipErrorNotInitialized) {
		return false;
	}
	return true;
}

/*!
	\brief Check how many CUDA-devices are present.
*/
int cudaDeviceFound(void) {

	int count;

	if(hipGetDeviceCount(&count) == hipSuccess) {
		return count;
	}
	return 0;
}

/*!
	\brief Read information about a CUDA-device.
*/
int cudaReadDeviceInfo(struct CZDeviceInfo *info, int num) {

	char name[256];
	unsigned int totalMem;
	int major;
	int minor;
	CUdevprop prop;
	int overlap;

	if(info == NULL)
		return -1;

	if(num >= cudaDeviceFound())
		return -1;

	if(hipDeviceGetName(name, 256, num) != hipSuccess)
		return -1;

	if(hipDeviceTotalMem(&totalMem, num) != hipSuccess)
		return -1;

	if(hipDeviceComputeCapability(&major, &minor, num) != hipSuccess)
		return -1;

	if(cuDeviceGetProperties(&prop, num) != hipSuccess)
		return -1;

	if(hipDeviceGetAttribute(&overlap, hipDeviceAttributeAsyncEngineCount, num) != hipSuccess)
		return -1;

	info->num = num;
	strcpy(info->deviceName, name);
	info->major = major;
	info->minor = minor;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.SIMDWidth = prop.SIMDWidth;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate;

	info->mem.totalGlobal = totalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstantMemory;
	info->mem.textureAlignment = prop.textureAlign;
	info->mem.gpuOverlap = overlap;

	return 0;
}
