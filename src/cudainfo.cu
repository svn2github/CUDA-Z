#include "hip/hip_runtime.h"
/*!	\file cudainfo.cu
	\brief CUDA information functions.
	\author Andriy Golovnya <andrew_golovnia@ukr.net> http://ag.embedded.org.ru/
	\url http://cuda-z.sf.net/ http://sf.net/projects/cuda-z/
	\license GPLv2 http://www.gnu.org/licenses/gpl-2.0.html
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <host_defines.h>
#include <string.h>

#if CUDA_VERSION < 5050
#error CUDA 1.x - 5.x are not supported any more! Please use CUDA Toolkit 5.5+ instead.
#endif

#include "log.h"
#include "cudainfo.h"

#if (defined(WIN64) || defined(_WIN64) || defined(__WIN64__)) || (defined(WIN32) || defined(_WIN32) || defined(__WIN32__) || defined(__NT__))
#define Q_OS_WIN
#elif defined(__linux__) || defined(__linux)
#define Q_OS_LINUX
#elif defined(__APPLE__) && defined(__GNUC__)
#define Q_OS_MAC
#else
#error Unknown/unsupported platform!
#endif

#define CZ_COPY_BUF_SIZE	(16 * (1 << 20))	/*!< Transfer buffer size. */
#define CZ_COPY_LOOPS_NUM	8			/*!< Number of loops to run transfer test to. */

#define CZ_CALC_BLOCK_LOOPS	16			/*!< Number of loops to run calculation loop. */
#define CZ_CALC_BLOCK_SIZE	256			/*!< Size of instruction block. */
#define CZ_CALC_BLOCK_NUM	16			/*!< Number of instruction blocks in loop. */
#define CZ_CALC_OPS_NUM		2			/*!< Number of operations per one loop. */
#define CZ_CALC_LOOPS_NUM	8			/*!< Number of loops to run performance test to. */

#define CZ_DEF_WARP_SIZE	32			/*!< Default warp size value. */
#define CZ_DEF_THREADS_MAX	512			/*!< Default max threads value value. */

#define CZ_VER_STR_LEN		256			/*!< Version string length. */

/*!	\brief Error handling of CUDA RT calls.
*/
#define CZ_CUDA_CALL(funcCall, errProc) \
	{ \
		hipError_t errCode; \
		if((errCode = (funcCall)) != hipSuccess) { \
			CZLog(CZLogLevelError, "CUDA Error: %08x %s", errCode, hipGetErrorString(errCode)); \
			errProc; \
		} \
	}

/*!	\brief Prototype of function \a hipDeviceGetAttribute().
*/
typedef hipError_t (CUDAAPI *cuDeviceGetAttribute_t)(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);

/*!	\brief Prototype of function \a hipInit().
*/
typedef hipError_t (CUDAAPI *cuInit_t)(unsigned int Flags);

/*!	\brief Pointer to function \a hipDeviceGetAttribute().
	This parameter is initializaed by CZCudaIsInit().
*/
static cuDeviceGetAttribute_t p_cuDeviceGetAttribute = NULL;

/*!	\brief Pointer to function \a hipInit().
	This parameter is initializaed by CZCudaIsInit().
*/
static cuInit_t p_cuInit = NULL;

/*!	\brief Driver version string.
*/
static char drvVersion[CZ_VER_STR_LEN] = "";

/*!	\brief Driver dll version.
*/
static int drvDllVer = 0;

/*!	\brief Driver dll version string.
*/
static char drvDllVerStr[CZ_VER_STR_LEN] = "";

/*!	\brief Runtime dll version.
*/
static int rtDllVer = 0;

/*!	\brief Runtime dll version string.
*/
static char rtDllVerStr[CZ_VER_STR_LEN] = "";

#if defined(Q_OS_WIN)
//#include <windows.h>
#define CZ_DLL_LIST_LEN		64			/*!< Process dll list length. */
#define CZ_DLL_BNAME_LEN	64			/*!< Dll base name length. */
#define CZ_DLL_FNAME_LEN	256			/*!< Dll file name length. */
#define CZ_DLL_FNAME		"nvcuda.dll"		/*!< CUDA dll file name. */

#ifdef __cplusplus
extern "C" {
#endif
#define WINAPI __stdcall
typedef void *HANDLE;
typedef void *HMODULE;
typedef void *HINSTANCE;
typedef HINSTANCE HMODULE;
typedef const char *LPCSTR;
typedef char *LPSTR;
typedef short DWORD, *LPDWORD;
typedef unsigned int UINT, *PUINT;
typedef void *LPVOID;
typedef const void *LPCVOID;
typedef bool BOOL;
typedef int (WINAPI *FARPROC)();
HMODULE WINAPI LoadLibraryA(__in LPCSTR lpLibFileName);
FARPROC WINAPI GetProcAddress(__in HMODULE hModule, __in LPCSTR lpProcName);
DWORD WINAPI GetFileVersionInfoSizeA(LPCSTR lptstrFilename, LPDWORD lpdwHandle);
BOOL WINAPI GetFileVersionInfoA(LPCSTR lptstrFilename, DWORD dwHandle, DWORD dwLen, LPVOID lpData);
BOOL WINAPI VerQueryValueA(LPCVOID pBlock, LPCSTR lpSubBlock, LPVOID *lplpBuffer, PUINT puLen);
HANDLE WINAPI GetCurrentProcess(void);
BOOL WINAPI EnumProcessModules(HANDLE hProcess, HMODULE *lphModule, DWORD cb, LPDWORD lpcbNeeded);
DWORD WINAPI GetModuleBaseNameA(HANDLE hProcess, HMODULE hModule, LPSTR lpBaseName, DWORD nSize);
DWORD WINAPI GetModuleFileNameA(HMODULE hModule, LPSTR lpFilename, DWORD nSize);
#ifdef __cplusplus
}
#endif

/*!	\brief Get version of dll library.
*/
static char *CZGetDllVersion(
	char *name,			/*!<[in] Name of dll file. */
	char *version			/*!<[out] Dll version buffer. */
) {
	DWORD dwVerInfoSize;
	DWORD dwVerHnd = 0;
	LPSTR lpstrVffInfo;
	LPSTR lpVersion = NULL;
	UINT uVersionLen = 0;

	dwVerInfoSize = GetFileVersionInfoSizeA(name, &dwVerHnd);
	if(!dwVerInfoSize) {
		return NULL;
	}

	lpstrVffInfo = (LPSTR)malloc(dwVerInfoSize);
	if(lpstrVffInfo == NULL) {
		return NULL;
	}

	if(!GetFileVersionInfoA(name, dwVerHnd, dwVerInfoSize, lpstrVffInfo)) {
		free(lpstrVffInfo);
		return NULL;
	}

	if(!VerQueryValueA(lpstrVffInfo, (LPSTR)"\\StringFileInfo\\040904E4\\FileVersion",
		(LPVOID*)&lpVersion, (UINT*)&uVersionLen)) {
		free(lpstrVffInfo);
		return NULL;
	}

	strncpy(version, lpVersion, CZ_VER_STR_LEN - 1);

	CZLog(CZLogLevelLow, "Version of %s is %s.", name, version);

	free(lpstrVffInfo);
	return version;
}

/*!	\brief Get description of dll library.
*/
static char *CZGetDllDescription(
	char *name,			/*!<[in] Name of dll file. */
	char *description		/*!<[out] Dll description buffer. */
) {
	DWORD dwVerInfoSize;
	DWORD dwVerHnd = 0;
	LPSTR lpstrVffInfo;
	LPSTR lpDescription = NULL;
	UINT uDescriptionLen = 0;

	dwVerInfoSize = GetFileVersionInfoSizeA(name, &dwVerHnd);
	if(!dwVerInfoSize) {
		return NULL;
	}

	lpstrVffInfo = (LPSTR)malloc(dwVerInfoSize);
	if(lpstrVffInfo == NULL) {
		return NULL;
	}

	if(!GetFileVersionInfoA(name, dwVerHnd, dwVerInfoSize, lpstrVffInfo)) {
		free(lpstrVffInfo);
		return NULL;
	}

	if(!VerQueryValueA(lpstrVffInfo, (LPSTR)"\\StringFileInfo\\040904E4\\FileDescription",
		(LPVOID*)&lpDescription, (UINT*)&uDescriptionLen)) {
		free(lpstrVffInfo);
		return NULL;
	}

	strncpy(description, lpDescription, CZ_VER_STR_LEN - 1);

	CZLog(CZLogLevelLow, "Description of %s is %s.", name, description);

	free(lpstrVffInfo);
	return description;
}

/*!	\brief Check if CUDA fully initialized.
	This function loads nvcuda.dll and finds functions \a hipInit()
	and \a hipDeviceGetAttribute().
	\return \a true in case of success, \a false in case of error.
*/
static bool CZCudaIsInit(void) {

	HINSTANCE hDll;
	HMODULE hModule[CZ_DLL_LIST_LEN];
	DWORD cbRet = 0;
	char description[CZ_VER_STR_LEN] = "";

	if((p_cuInit == NULL) || (p_cuDeviceGetAttribute == NULL)) {

		hDll = LoadLibraryA(CZ_DLL_FNAME);
		if(hDll == NULL) {
			return false;
		}

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)GetProcAddress(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
			return false;
		}

		p_cuInit = (cuInit_t)GetProcAddress(hDll, "hipInit");
		if(p_cuInit == NULL) {
			return false;
		}

		CZGetDllVersion(CZ_DLL_FNAME, drvDllVerStr);

		if(CZGetDllDescription(CZ_DLL_FNAME, description) != NULL) {
			char *p = NULL;
			char *version = "version";
			strlwr(description);
			if((p = strstr(description, version)) != NULL) {
				p += strlen(version);
				while(*p == ' ')
					p++;
				strncpy(drvVersion, p, CZ_VER_STR_LEN - 1);
				p = drvVersion + strlen(drvVersion) - 1;
				while(*p == ' ') {
					*p = 0;
					p--;
				}
			}
		}

		if(EnumProcessModules(GetCurrentProcess(), hModule, sizeof(hModule), &cbRet) == true) {
			UINT i;
			char bname[CZ_DLL_BNAME_LEN];
			char fname[CZ_DLL_FNAME_LEN];
			for(i = 0; i < (cbRet / sizeof(HMODULE)); i++) {
				bname[0] = 0;
				fname[0] = 0;
				GetModuleBaseNameA(GetCurrentProcess(), hModule[i], bname, CZ_DLL_BNAME_LEN - 1);
				strlwr(bname);
				if(strstr(bname, "cudart") != NULL) {
					GetModuleFileNameA(hModule[i], fname, CZ_DLL_FNAME_LEN - 1);
					CZGetDllVersion(fname, rtDllVerStr);
					break;
				}
			}
		}
	}

	return true;
}

#elif defined(Q_OS_LINUX)
#include <dlfcn.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#define CZ_FILE_STR_LEN		256			/*!< Version file string length. */
#define CZ_VER_FILE_NAME	"/proc/driver/nvidia/version"	/*!< Driver version file name. */
#define CZ_PROC_MAP_NAME	"/proc/self/maps"	/*!< Process memory map file. */
#define CZ_DLL_FNAME		"libcuda.so"		/*!< CUDA dll file name. */
#define CZ_DLL_FNAME_RT		"libcudart.so"		/*!< CUDA RT dll file name. */
#define CZ_LD_SO_CONF		"/etc/ld.do.conf"	/*!< ld.so configuration file. */
#define CZ_LD_SO_DIR		"/etc/ld.do.conf.d/"	/*!< ld.so configuration directory. */
#define CZ_LD_SO_LINE_MAX	100			/*!< ld.so configuration line length. */

/*!	\brief Get version of shared library.
*/
static char *CZGetSoVersion(
	char *name,			/*!<[in] Name of so file. E.g. "libcuda.so". */
	char *version			/*!<[out] Library version buffer. */
) {
	FILE *fp = NULL;
	char str[CZ_FILE_STR_LEN];
	int found = 0;

	fp = fopen(CZ_PROC_MAP_NAME, "r");
	if(fp == NULL) {
		return NULL;
	}

	while(fgets(str, CZ_FILE_STR_LEN - 1, fp) != NULL) {
		if(strstr(str, name) != NULL) {
			char *p = NULL;
			char fname[CZ_FILE_STR_LEN];

			p = str + strlen(str) - 1;
			while((p >= str) && ((*p == ' ') || (*p == '\n') || (*p == '\r') || (*p == '\t') || (*p == 0))) {
				*p = 0;
				p--;
			}

			while((p >= str) && ((*p != ' ') && (*p != '\n') && (*p != '\r') && (*p != '\t') && (*p != 0))) {
				p--;
			}

			strncpy(fname, p, CZ_FILE_STR_LEN - 1);
			p = basename(fname);
			if(p == NULL)
				continue;

			if(strstr(p, name) != p) {
				continue;
			}

			p = p + strlen(name);
			if(*p != '.')
				continue;

			strncpy(version, p + 1, CZ_VER_STR_LEN - 1);

			found++;
			break;
		}
	}

	fclose(fp);

	if(found) {
		CZLog(CZLogLevelLow, "Version of %s is %s.", name, version);
		return version;
	} else {
		return NULL;
	}
}

/*!	\brief Check if CUDA fully initialized.
	This function loads libcuda.so and finds functions \a hipInit()
	and \a hipDeviceGetAttribute().
	\return \a true in case of success, \a false in case of error.
*/
static bool CZCudaIsInit(void) {

	void *hDll = NULL;

	if((p_cuInit == NULL) || (p_cuDeviceGetAttribute == NULL)) {

		if(hDll == NULL) {
			hDll = dlopen(CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib32/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib32/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib64/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib64/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib128/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib128/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		// Try to check ld.so.conf & Co
		if(hDll == NULL) {
			char buf[CZ_LD_SO_LINE_MAX + sizeof(CZ_DLL_FNAME) + 1];
			char *p;
			FILE *f;

			f = popen("cat " CZ_LD_SO_CONF " " CZ_LD_SO_DIR "/*", "r");
			if(f != NULL) {
				while(fgets(buf, CZ_LD_SO_LINE_MAX, f) != NULL) {
					if((p = strchr(buf, '\n')) != NULL) *p = 0;
					if((p = strchr(buf, '#')) != NULL) *p = 0;
					if(strlen(buf) > 0) {
						strcat(buf, "/" CZ_DLL_FNAME);
						if(hDll == NULL) {
							hDll = dlopen(buf, RTLD_LAZY);
						}
					}
				}
				pclose(f);
			}
		}

		if(hDll == NULL) {
			CZLog(CZLogLevelError, "Can't load CUDA driver.");
			return false;
		}

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)dlsym(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
			return false;
		}

		p_cuInit = (cuInit_t)dlsym(hDll, "hipInit");
		if(p_cuInit == NULL) {
			return false;
		}

		CZGetSoVersion(CZ_DLL_FNAME, drvDllVerStr);
		CZGetSoVersion(CZ_DLL_FNAME_RT, rtDllVerStr);

		if(access(CZ_VER_FILE_NAME, R_OK) == 0) {
			FILE *fp = NULL;
			char str[CZ_FILE_STR_LEN];
			fp = fopen(CZ_VER_FILE_NAME, "r");
			if(fp != NULL) {
				while(fgets(str, CZ_FILE_STR_LEN - 1, fp) != NULL) {
					char *p = NULL;
					char *kernel_module = "Kernel Module";
					if((p = strstr(str, kernel_module)) != NULL) {
						p += strlen(kernel_module);
						while(*p == ' ')
							p++;
						strncpy(drvVersion, p, CZ_VER_STR_LEN - 1);
						p = drvVersion;
						while((*p != ' ') && (*p != '\n') && (*p != '\r') && (*p != '\t') && (*p != 0)) {
							p++;
						}
						*p = 0;
						break;
					}
				}
				fclose(fp);
			}
		}

	}
	return true;
}

#elif defined(Q_OS_MAC)
#include <dlfcn.h>
#include <stdio.h>
#include "plist.h"
#define CZ_FILE_STR_LEN		256			/*!< Version file string length. */
#define CZ_PLIST_PATH		"/Contents/Info.plist"	/*!< Path to Info.plist inside of kext/app. */
#define CZ_KEXT_SYSTEM_PATH	"/System/Library/Extensions/"	/*!< Main kext path. */
#define CZ_KEXT_EXTRA_PATH	"/Extra/Extensions/"	/*!< Alternative kext path. */
#define CZ_PLIST_GETINFOSTR	"CFBundleGetInfoString"	/*!< Most informative property name. */
#define CZ_PLIST_SHORTVERSTR	"CFBundleShortVersionString"	/*!< Less informative property name. */
#define CZ_DLL_FNAME		"libcuda.dylib"		/*!< CUDA dll file name. */

/*!	\brief Get version of Kext driver.
*/
static char *CZGetKextVersion(
	char *name,			/*!<[in] Name of kext file. E.g. "GeForce". */
	char *version			/*!<[out] Kext version buffer. */
) {
	char plist[CZ_FILE_STR_LEN];
	char str[CZ_FILE_STR_LEN];
	char *p;

	sprintf(plist, CZ_KEXT_SYSTEM_PATH "%s.kext" CZ_PLIST_PATH, name);

	if(CZPlistGet(plist, CZ_PLIST_GETINFOSTR, str, sizeof(str)) == 0) {
		p = strstr(str, name);
		if(p != NULL) {
			p = p + strlen(name);
			while(*p == ' ')
				p++;
		} else {
			p = str;
		}
		strcpy(version, p);
		return version;
	}

	if(CZPlistGet(plist, CZ_PLIST_SHORTVERSTR, str, sizeof(str)) == 0) {
		p = strstr(str, name);
		if(p != NULL) {
			p = p + strlen(name);
			while(*p == ' ')
				p++;
		} else {
			p = str;
		}
		strcpy(version, p);
		return version;
	}

	sprintf(plist, CZ_KEXT_EXTRA_PATH "%s.kext" CZ_PLIST_PATH, name);

	if(CZPlistGet(plist, CZ_PLIST_GETINFOSTR, str, sizeof(str)) == 0) {
		p = strstr(str, name);
		if(p != NULL) {
			p = p + strlen(name);
			while(*p == ' ')
				p++;
		} else {
			p = str;
		}
		strcpy(version, p);
		return version;
	}

	if(CZPlistGet(plist, CZ_PLIST_SHORTVERSTR, str, sizeof(str)) == 0) {
		p = strstr(str, name);
		if(p != NULL) {
			p = p + strlen(name);
			while(*p == ' ')
				p++;
		} else {
			p = str;
		}
		strcpy(version, p);
		return version;
	}

	return NULL;
}

/*!	\brief Check if CUDA fully initialized.
	This function loads libcuda.dylib and finds functions \a hipInit()
	and \a hipDeviceGetAttribute().
	\return \a true in case of success, \a false in case of error.
*/
static bool CZCudaIsInit(void) {
	void *hDll = NULL;

	if((p_cuInit == NULL) || (p_cuDeviceGetAttribute == NULL)) {

		if(hDll == NULL) {
			hDll = dlopen(CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("@rpath/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("@executable_path/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/local/cuda/lib/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			CZLog(CZLogLevelError, "Can't load CUDA driver.");
			return false;
		}

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)dlsym(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
			return false;
		}

		p_cuInit = (cuInit_t)dlsym(hDll, "hipInit");
		if(p_cuInit == NULL) {
			return false;
		}

		CZGetKextVersion("GeForce", drvVersion);
	}
	return true;
}

#else//!Q_OS_WIN && !Q_OS_LINUX && !Q_OS_MAC
#error Function CZCudaIsInit() is not implemented for your platform!
#endif//Q_OS_WIN

/*!	\brief Check if CUDA is present here.
*/
bool CZCudaCheck(void) {

	if(!CZCudaIsInit())
		return false;

	if(p_cuInit(0) == hipErrorNotInitialized) {
		return false;
	}

	CZ_CUDA_CALL(hipDriverGetVersion(&drvDllVer),
		drvDllVer = 0);

	CZLog(CZLogLevelLow, "Driver version %d.", drvDllVer);

	CZ_CUDA_CALL(hipRuntimeGetVersion(&rtDllVer),
		rtDllVer = 0);

	CZLog(CZLogLevelLow, "Runtime version %d.", rtDllVer);

	return true;
}

/*!	\brief Check how many CUDA-devices are present.
	\return number of CUDA-devices in case of success, \a 0 if no CUDA-devies were found.
*/
int CZCudaDeviceFound(void) {

	int count;

	CZ_CUDA_CALL(hipGetDeviceCount(&count),
		return 0);

	return count;
}

/*!	\def ConvertSMVer2Cores(major, minor)
	\brief Get number of CUDA cores per multiprocessor.
	\arg[in] major GPU Architecture major version.
	\arg[in] minor GPU Architecture minor version.
	\returns 0 if GPU Architecture is unknown, or number of CUDA cores per multiprocessor.
*/
#define ConvertSMVer2Cores(major, minor) \
	(((major) == 1)? ( \
		((minor) == 0)? 8: /* G80*/ \
		((minor) == 1)? 8: /* G8x */ \
		((minor) == 2)? 8: /* G9x */ \
		((minor) == 3)? 8: /* GT200 */ \
		0): \
	((major) == 2)? ( \
		((minor) == 0)? 32: /* GF100 */ \
		((minor) == 1)? 48: /* GF10x */ \
		0): \
	((major) == 3)? ( \
		((minor) == 0)? 192: /* GK10x */ \
		((minor) == 5)? 192: /* GK11x */ \
		0): \
	0)

/*!	\def COMPILE_ASSERT(cond)
	\arg[in] cond Static condition.
	\brief Compile time assert() for constant conditions.
*/
#define COMPILE_ASSERT(cond)	{typedef char compile_assert_error[(cond)? 1: -1];}

/*!	\brief Read information about a CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaReadDeviceInfo(
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int num				/*!<[in] Number (index) of CUDA-device. */
) {
	hipDeviceProp_t prop;
//	int ecc;

	COMPILE_ASSERT(ConvertSMVer2Cores(0, 0) == 0);
	COMPILE_ASSERT(ConvertSMVer2Cores(1, 0) == 8);
	COMPILE_ASSERT(ConvertSMVer2Cores(1, 1) == 8);
	COMPILE_ASSERT(ConvertSMVer2Cores(1, 2) == 8);
	COMPILE_ASSERT(ConvertSMVer2Cores(1, 3) == 8);
	COMPILE_ASSERT(ConvertSMVer2Cores(1, 4) == 0);
	COMPILE_ASSERT(ConvertSMVer2Cores(2, 0) == 32);
	COMPILE_ASSERT(ConvertSMVer2Cores(2, 1) == 48);
	COMPILE_ASSERT(ConvertSMVer2Cores(2, 2) == 0);
	COMPILE_ASSERT(ConvertSMVer2Cores(3, 0) == 192);
	COMPILE_ASSERT(ConvertSMVer2Cores(3, 1) == 0);
	COMPILE_ASSERT(ConvertSMVer2Cores(3, 5) == 192);
	COMPILE_ASSERT(ConvertSMVer2Cores(3, 6) == 0);
	COMPILE_ASSERT(ConvertSMVer2Cores(4, 0) == 0);

	if(info == NULL)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(num >= CZCudaDeviceFound())
		return -1;

	CZ_CUDA_CALL(hipGetDeviceProperties(&prop, num),
		return -1);

	info->num = num;
	strcpy(info->deviceName, prop.name);
	info->major = prop.major;
	info->minor = prop.minor;
	info->drvVersion = drvVersion;
	info->drvDllVer = drvDllVer;
	info->drvDllVerStr = drvDllVerStr;
	info->rtDllVer = rtDllVer;
	info->rtDllVerStr = rtDllVerStr;
	info->tccDriver = prop.tccDriver;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.SIMDWidth = prop.warpSize;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate;
	info->core.muliProcCount = prop.multiProcessorCount;
	info->core.watchdogEnabled = prop.kernelExecTimeoutEnabled;
	info->core.integratedGpu = prop.integrated;
	info->core.concurrentKernels = prop.concurrentKernels;
	info->core.computeMode =
		(prop.computeMode == hipComputeModeDefault)? CZComputeModeDefault:
		(prop.computeMode == hipComputeModeExclusive)? CZComputeModeExclusive:
		(prop.computeMode == hipComputeModeProhibited)? CZComputeModeProhibited:
		CZComputeModeUnknown;
	info->core.pciBusID = prop.pciBusID;
	info->core.pciDeviceID = prop.pciDeviceID;
	info->core.pciDomainID = prop.pciDomainID;
	info->core.maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
	info->core.cudaCores = ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount;
	info->core.streamPrioritiesSupported = prop.streamPrioritiesSupported;

	info->mem.totalGlobal = prop.totalGlobalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstMem;
	info->mem.textureAlignment = prop.textureAlignment;
	info->mem.texture1D[0] = prop.maxTexture1D;
	info->mem.texture2D[0] = prop.maxTexture2D[0];
	info->mem.texture2D[1] = prop.maxTexture2D[1];
	info->mem.texture3D[0] = prop.maxTexture3D[0];
	info->mem.texture3D[1] = prop.maxTexture3D[1];
	info->mem.texture3D[2] = prop.maxTexture3D[2];
	info->mem.gpuOverlap = prop.deviceOverlap;
	info->mem.mapHostMemory = prop.canMapHostMemory;
        info->mem.errorCorrection = prop.ECCEnabled;
	info->mem.asyncEngineCount = prop.asyncEngineCount;
	info->mem.unifiedAddressing = prop.unifiedAddressing;
	info->mem.memoryClockRate = prop.memoryClockRate;
	info->mem.memoryBusWidth = prop.memoryBusWidth;
	info->mem.l2CacheSize = prop.l2CacheSize;

/*	if(p_cuDeviceGetAttribute(&ecc, hipDeviceAttributeEccEnabled, num) != hipSuccess)
		return -1;
	info->mem.errorCorrection = ecc;*/

	return 0;
}

/*!	\brief Local service data structure for bandwith calulations.
*/
struct CZDeviceInfoBandLocalData {
	void		*memHostPage;	/*!< Pageable host memory. */
	void		*memHostPin;	/*!< Pinned host memory. */
	void		*memDevice1;	/*!< Device memory buffer 1. */
	void		*memDevice2;	/*!< Device memory buffer 2. */
};

/*!	\brief Set device for current thread.
*/
int CZCudaCalcDeviceSelect(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	CZLog(CZLogLevelLow, "Selecting %s.", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return -1);

	return 0;
}

/*!	\brief Allocate buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthAlloc(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	if(info->band.localData == NULL) {

		CZLog(CZLogLevelLow, "Alloc local buffers for %s.", info->deviceName);

		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(*lData));
		if(lData == NULL) {
			return -1;
		}

		CZLog(CZLogLevelLow, "Alloc host pageable for %s.", info->deviceName);

		lData->memHostPage = (void*)malloc(CZ_COPY_BUF_SIZE);
		if(lData->memHostPage == NULL) {
			free(lData);
			return -1;
		}

		CZLog(CZLogLevelLow, "Host pageable is at 0x%08X.", lData->memHostPage);

		CZLog(CZLogLevelLow, "Alloc host pinned for %s.", info->deviceName);

		CZ_CUDA_CALL(hipHostMalloc((void**)&lData->memHostPin, CZ_COPY_BUF_SIZE),
			free(lData->memHostPage);
			free(lData);
			return -1);

		CZLog(CZLogLevelLow, "Host pinned is at 0x%08X.", lData->memHostPin);

		CZLog(CZLogLevelLow, "Alloc device buffer 1 for %s.", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice1, CZ_COPY_BUF_SIZE),
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		CZLog(CZLogLevelLow, "Device buffer 1 is at 0x%08X.", lData->memDevice1);

		CZLog(CZLogLevelLow, "Alloc device buffer 2 for %s.", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice2, CZ_COPY_BUF_SIZE),
			hipFree(lData->memDevice1);
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		CZLog(CZLogLevelLow, "Device buffer 2 is at 0x%08X.", lData->memDevice2);

		info->band.localData = (void*)lData;
	}

	return 0;
}

/*!	\brief Free buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthFree(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	if(lData != NULL) {

		CZLog(CZLogLevelLow, "Free host pageable for %s.", info->deviceName);

		if(lData->memHostPage != NULL)
			free(lData->memHostPage);

		CZLog(CZLogLevelLow, "Free host pinned for %s.", info->deviceName);

		if(lData->memHostPin != NULL)
			hipHostFree(lData->memHostPin);

		CZLog(CZLogLevelLow, "Free device buffer 1 for %s.", info->deviceName);

		if(lData->memDevice1 != NULL)
			hipFree(lData->memDevice1);

		CZLog(CZLogLevelLow, "Free device buffer 2 for %s.", info->deviceName);

		if(lData->memDevice2 != NULL)
			hipFree(lData->memDevice2);

		CZLog(CZLogLevelLow, "Free local buffers for %s.", info->deviceName);

		free(lData);
	}
	info->band.localData = NULL;

	return 0;
}

/*!	\brief Reset results of bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthReset(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->band.copyHDPage = 0;
	info->band.copyHDPin = 0;
	info->band.copyDHPage = 0;
	info->band.copyDHPin = 0;
	info->band.copyDD = 0;

	return 0;
}

#define CZ_COPY_MODE_H2D	0	/*!< Host to device data copy mode. */
#define CZ_COPY_MODE_D2H	1	/*!< Device to host data copy mode. */
#define CZ_COPY_MODE_D2D	2	/*!< Device to device data copy mode. */

/*!	\brief Run data transfer bandwidth tests.
	\return \a 0 in case of success, \a other is value in KiB/s.
*/
static float CZCudaCalcDeviceBandwidthTestCommon (
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int mode,			/*!<[in] Run bandwidth test in one of modes. */
	int pinned			/*!<[in] Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKiBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice1;
	void *memDevice2;
	int i;

	if(info == NULL)
		return 0;

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice1 = lData->memDevice1;
	memDevice2 = lData->memDevice2;

	CZLog(CZLogLevelLow, "Starting %s test (%s) on %s.",
		(mode == CZ_COPY_MODE_H2D)? "host to device":
		(mode == CZ_COPY_MODE_D2H)? "device to host":
		(mode == CZ_COPY_MODE_D2D)? "device to device": "unknown",
		pinned? "pinned": "pageable",
		info->deviceName);

	for(i = 0; i < CZ_COPY_LOOPS_NUM; i++) {

		float loopMs = 0.0;

		CZ_CUDA_CALL(hipEventRecord(start, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		switch(mode) {
		case CZ_COPY_MODE_H2D:
			CZ_CUDA_CALL(hipMemcpy(memDevice1, memHost, CZ_COPY_BUF_SIZE, hipMemcpyHostToDevice),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		case CZ_COPY_MODE_D2H:
			CZ_CUDA_CALL(hipMemcpy(memHost, memDevice2, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToHost),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		case CZ_COPY_MODE_D2D:
			CZ_CUDA_CALL(hipMemcpy(memDevice2, memDevice1, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToDevice),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		default: // WTF!
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}

		CZ_CUDA_CALL(hipEventRecord(stop, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventSynchronize(stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventElapsedTime(&loopMs, start, stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		timeMs += loopMs;
	}

	CZLog(CZLogLevelLow, "Test complete in %f ms.", timeMs);

	bandwidthKiBs = (
		1000 *
		(float)CZ_COPY_BUF_SIZE *
		(float)CZ_COPY_LOOPS_NUM
	) / (
		timeMs *
		(float)(1 << 10)
	);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return bandwidthKiBs;
}

/*!	\brief Run several bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDeviceBandwidthTest(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	info->band.copyHDPage = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_H2D, 0);
	info->band.copyHDPin = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_H2D, 1);
	info->band.copyDHPage = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2H, 0);
	info->band.copyDHPin = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2H, 1);
	info->band.copyDD = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2D, 0);

	return 0;
}

/*!	\brief Prepare buffers bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaPrepareDevice(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	return 0;
}

/*!	\brief Calculate bandwidth information about CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaCalcDeviceBandwidth(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthReset(info) != 0)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	if(CZCudaCalcDeviceBandwidthTest(info) != 0)
		return -1;

	return 0;
}

/*!	\brief Cleanup after test and bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaCleanDevice(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthFree(info) != 0)
		return -1;

	return 0;
}

/*!	\brief Reset results of preformance calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static int CZCudaCalcDevicePerformanceReset(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->perf.calcFloat = 0;
	info->perf.calcDouble = 0;
	info->perf.calcInteger32 = 0;
	info->perf.calcInteger24 = 0;

	return 0;
}

/*!	\brief 16 MAD instructions for float point test.
*/
#define CZ_CALC_FMAD_16(a, b) \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \

/*!	\brief 256 MAD instructions for float point test.
*/
#define CZ_CALC_FMAD_256(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \

/*!	\brief 16 DMAD instructions for double-precision test.
*/
#define CZ_CALC_DFMAD_16(a, b) \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \

/*	a = fma(b, a, b); b = fma(a, b, a); a = fma(b, a, b); b = fma(a, b, a); \
	a = fma(b, a, b); b = fma(a, b, a); a = fma(b, a, b); b = fma(a, b, a); \
	a = fma(b, a, b); b = fma(a, b, a); a = fma(b, a, b); b = fma(a, b, a); \
	a = fma(b, a, b); b = fma(a, b, a); a = fma(b, a, b); b = fma(a, b, a); \*/

/*!	\brief 256 MAD instructions for float point test.
*/
#define CZ_CALC_DFMAD_256(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \

/*!	\brief 16 MAD instructions for 32-bit integer test.
*/
#define CZ_CALC_IMAD32_16(a, b) \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \
	a = b * a + b; b = a * b + a; a = b * a + b; b = a * b + a; \

/*!	\brief 256 MAD instructions for 32-bit integer test.
*/
#define CZ_CALC_IMAD32_256(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \

/*!	\brief 16 MAD instructions for 24-bit integer test.
*/
#define CZ_CALC_IMAD24_16(a, b) \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \
	a = __umul24(b, a) + b; b = __umul24(a, b) + a; \

/*!	\brief 256 MAD instructions for 24-bit integer test.
*/
#define CZ_CALC_IMAD24_256(a, b) \
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\

#define CZ_CALC_MODE_FLOAT	0	/*!< Single-precision float point test mode. */
#define CZ_CALC_MODE_DOUBLE	1	/*!< Double-precision float point test mode. */
#define CZ_CALC_MODE_INTEGER32	2	/*!< 32-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER24	3	/*!< 24-bit integer test mode. */

/*!	\brief GPU code for float point test.
*/
__global__ void CZCudaCalcKernelFloat(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float *arr = (float*)buf;
	float val1 = index;
	float val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for double-precision test.
*/
__global__ void CZCudaCalcKernelDouble(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	double *arr = (double*)buf;
	double val1 = index;
	double val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for 32-bit integer test.
*/
__global__ void CZCudaCalcKernelInteger32(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for 24-bit integer test.
*/
__global__ void CZCudaCalcKernelInteger24(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief Run GPU calculation performace tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
static float CZCudaCalcDevicePerformanceTest(
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int mode			/*!<[in] Run performance test in one of modes. */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float performanceKOPs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	int blocksNum = info->heavyMode? info->core.muliProcCount: 1;
	int i;

	if(info == NULL)
		return 0;

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	int threadsNum = info->core.maxThreadsPerBlock;
	if(threadsNum == 0) {
		int warpSize = info->core.SIMDWidth;
		if(warpSize == 0)
			warpSize = CZ_DEF_WARP_SIZE;
		threadsNum = warpSize * 2;
		if(threadsNum > CZ_DEF_THREADS_MAX)
			threadsNum = CZ_DEF_THREADS_MAX;
	}

	CZLog(CZLogLevelLow, "Starting %s test on %s on %d block(s) %d thread(s) each.",
		(mode == CZ_CALC_MODE_FLOAT)? "single-precision float":
		(mode == CZ_CALC_MODE_DOUBLE)? "double-precision float":
		(mode == CZ_CALC_MODE_INTEGER32)? "32-bit integer":
		(mode == CZ_CALC_MODE_INTEGER24)? "24-bit integer": "unknown",
		info->deviceName,
		blocksNum,
		threadsNum);

	for(i = 0; i < CZ_CALC_LOOPS_NUM; i++) {

		float loopMs = 0.0;

		CZ_CUDA_CALL(hipEventRecord(start, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		switch(mode) {
		case CZ_CALC_MODE_FLOAT:
			CZCudaCalcKernelFloat<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_DOUBLE:
			CZCudaCalcKernelDouble<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER32:
			CZCudaCalcKernelInteger32<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER24:
			CZCudaCalcKernelInteger24<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		default: // WTF!
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}

		CZ_CUDA_CALL(hipGetLastError(),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventRecord(stop, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventSynchronize(stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventElapsedTime(&loopMs, start, stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		timeMs += loopMs;
	}

	CZLog(CZLogLevelLow, "Test complete in %f ms.", timeMs);

	performanceKOPs = (
		(float)info->core.muliProcCount *
		(float)CZ_CALC_LOOPS_NUM *
		(float)threadsNum *
		(float)CZ_CALC_BLOCK_LOOPS *
		(float)CZ_CALC_OPS_NUM *
		(float)CZ_CALC_BLOCK_SIZE *
		(float)CZ_CALC_BLOCK_NUM
	) / (float)timeMs;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return performanceKOPs;
}

/*!	\brief Calculate performance information about CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaCalcDevicePerformance(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDevicePerformanceReset(info) != 0)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	info->perf.calcFloat = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_FLOAT);
	if(((info->major > 1)) ||
		((info->major == 1) && (info->minor >= 3)))
		info->perf.calcDouble = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_DOUBLE);
	info->perf.calcInteger32 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER32);
	info->perf.calcInteger24 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER24);

	return 0;
}
