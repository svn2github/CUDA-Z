/*!
	\file cudainfo.cu
	\brief CUDA information functions.
	\author AG
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <windows.h>

#include <QDebug>

#include "cudainfo.h"

#define CZ_BAND_BUF_SIZE	(10 * (1 << 20))	/*!< Transfer buffer size. */
#define CZ_BAND_LOOPS_NUM	(10)			/*!< Number of loops to run transfer test to. */

/*!
	\brief Error handling of CUDA RT calls.
*/
#define CZ_CUDA_CALL(funcCall, errProc) \
	{ \
		hipError_t errCode; \
		if((errCode = (funcCall)) != hipSuccess) { \
			printf("CUDA Error: %s\n", hipGetErrorString(errCode)); \
			errProc; \
		} \
	}

/*!
	\brief Prototype of function \a hipDeviceGetAttribute().
*/
typedef hipError_t (CUDAAPI *cuDeviceGetAttribute_t)(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);

/*!
	\brief Prototype of function \a hipInit().
*/
typedef hipError_t (CUDAAPI *cuInit_t)(unsigned int Flags);

/*!
	\brief Pointer to function \a hipDeviceGetAttribute().
	This parameter is initializaed by #cudaIsInit().
*/
static cuDeviceGetAttribute_t p_cuDeviceGetAttribute = NULL;

/*!
	\brief Pointer to function \a hipInit().
	This parameter is initializaed by #cudaIsInit().
*/
static cuInit_t p_cuInit = NULL;

/*!
	\brief Check if CUDa fully initialized.
	This function loads nvcuda.dll and finds function \a hipDeviceGetAttribute.
	\return \a true in case of success, \a false in case of error.
*/
static bool cudaIsInit(void) {

	HINSTANCE hDll;

	if((p_cuInit == NULL) || (p_cuDeviceGetAttribute == NULL)) {

		hDll = LoadLibrary(L"nvcuda.dll");
		if(hDll == NULL) {
			return false;
		}

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)GetProcAddress(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
			return false;
		}

		p_cuInit = (cuInit_t)GetProcAddress(hDll, "hipInit");
		if(p_cuInit == NULL) {
			return false;
		}
	}

	return true;
}

/*!
	\brief Check if CUDA is present here.
*/
bool cudaCheck(void) {

	if(!cudaIsInit())
		return false;

	if(p_cuInit(0) == hipErrorNotInitialized) {
		return false;
	}

	return true;
}

/*!
	\brief Check how many CUDA-devices are present.
	\return number of CUDA-devices in case of success, \a 0 if no CUDA-devies were found.
*/
int cudaDeviceFound(void) {

	int count;

	CZ_CUDA_CALL(hipGetDeviceCount(&count),
		return 0);
	return count;
}

/*!
	\brief Read information about a CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaReadDeviceInfo(
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int num				/*!< Number (index) of CUDA-device. */
) {
	hipDeviceProp_t prop;
	int overlap;

	if(info == NULL)
		return -1;

	if(!cudaIsInit())
		return -1;

	if(num >= cudaDeviceFound())
		return -1;

	CZ_CUDA_CALL(hipGetDeviceProperties(&prop, num),
		return -1);

	if(p_cuDeviceGetAttribute(&overlap, hipDeviceAttributeAsyncEngineCount, num) != hipSuccess)
		return -1;

	info->num = num;
	strcpy(info->deviceName, prop.name);
	info->major = prop.major;
	info->minor = prop.minor;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.SIMDWidth = prop.warpSize;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate;

	info->mem.totalGlobal = prop.totalGlobalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstMem;
	info->mem.textureAlignment = prop.textureAlignment;
	info->mem.gpuOverlap = overlap;

	return 0;
}

/*!
	\brief Local service data structure for bandwith calulations.
*/
struct CZDeviceInfoBandLocalData {
	void		*memHostPage;	/*!< Pageable host memory. */
	void		*memHostPin;	/*!< Pinned host memory. */
	void		*memDevice1;	/*!< Device memory buffer 1. */
	void		*memDevice2;	/*!< Device memory buffer 2. */
};

/*!
	\brief Allocate buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthAlloc(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	if(info->band.localData == NULL) {

		printf("Selecting %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipSetDevice(info->num),
			return -1);

		printf("Alloc local buffers for %s.\n", info->deviceName);

		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(*lData));
		if(lData == NULL) {
			return -1;
		}

		printf("Alloc host pageable for %s.\n", info->deviceName);

		lData->memHostPage = (void*)malloc(CZ_BAND_BUF_SIZE);
		if(lData->memHostPage == NULL) {
			free(lData);
			return -1;
		}

		printf("Host pageable is at 0x%08X.\n", lData->memHostPage);

		printf("Alloc host pinned for %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipHostMalloc((void**)&lData->memHostPin, CZ_BAND_BUF_SIZE),
			free(lData->memHostPage);
			free(lData);
			return -1);

		printf("Host pinned is at 0x%08X.\n", lData->memHostPin);

		printf("Alloc device buffer 1 for %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice1, CZ_BAND_BUF_SIZE),
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		printf("Device buffer 1 is at 0x%08X.\n", lData->memDevice1);

		printf("Alloc device buffer 2 for %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice2, CZ_BAND_BUF_SIZE),
			hipFree(lData->memDevice1);
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		printf("Device buffer 2 is at 0x%08X.\n", lData->memDevice2);

		info->band.localData = (void*)lData;
	}

	return 0;
}

/*!
	\brief Free buffers for bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthFree(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	if(lData != NULL) {

		printf("Selecting %s.\n", info->deviceName);

		CZ_CUDA_CALL(hipSetDevice(info->num),
			return -1);

		printf("Free host pageable for %s.\n", info->deviceName);

		if(lData->memHostPage != NULL)
			free(lData->memHostPage);

		printf("Free host pinned for %s.\n", info->deviceName);

		if(lData->memHostPin != NULL)
			hipHostFree(lData->memHostPin);

		printf("Free device buffer 1 for %s.\n", info->deviceName);

		if(lData->memDevice1 != NULL)
			hipFree(lData->memDevice1);

		printf("Free device buffer 2 for %s.\n", info->deviceName);

		if(lData->memDevice2 != NULL)
			hipFree(lData->memDevice2);

		printf("Free local buffers for %s.\n", info->deviceName);

		free(lData);
	}
	info->band.localData = NULL;

	return 0;
}

/*!
	\brief Reset results of bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthReset(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->band.copyHDPage = 0;
	info->band.copyHDPin = 0;
	info->band.copyDHPage = 0;
	info->band.copyDHPin = 0;
	info->band.copyDD = 0;

	return 0;
}

/*!
	\brief Run host to device data transfer bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthTestHD (
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int pinned			/*!< Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice = lData->memDevice1;

	printf("Starting H(0x%08X)->D(0x%08X) test (%s) on %s.\n",
		memHost, memDevice,
		pinned? "pinned": "pageable",
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	for(i = 0; i < CZ_BAND_LOOPS_NUM; i++) {
		CZ_CUDA_CALL(hipMemcpy(memDevice, memHost, CZ_BAND_BUF_SIZE, hipMemcpyHostToDevice),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (1000 * (float)CZ_BAND_BUF_SIZE * (float)CZ_BAND_LOOPS_NUM) / (timeMs * (float)(1 << 10));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run device to host data transfer bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthTestDH (
	struct CZDeviceInfo *info,	/*!< CUDA-device information. */
	int pinned			/*!< Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice = lData->memDevice2;

	printf("Starting D(0x%08X)->H(0x%08X) test (%s) on %s.\n",
		memDevice, memHost,
		pinned? "pinned": "pageable",
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	for(i = 0; i < CZ_BAND_LOOPS_NUM; i++) {
		CZ_CUDA_CALL(hipMemcpy(memDevice, memHost, CZ_BAND_BUF_SIZE, hipMemcpyHostToDevice),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (1000 * (float)CZ_BAND_BUF_SIZE * (float)CZ_BAND_LOOPS_NUM) / (timeMs * (float)(1 << 10));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run device to device data transfer bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthTestDD (
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memDevice1;
	void *memDevice2;
	int i;

	if(info == NULL)
		return 0;

	printf("Selecting %s.\n", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memDevice1 = lData->memDevice1;
	memDevice2 = lData->memDevice2;

	printf("Starting D(0x%08X)->D(0x%08X) test on %s.\n",
		memDevice1, memDevice2,
		info->deviceName);

	CZ_CUDA_CALL(hipEventRecord(start, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	for(i = 0; i < CZ_BAND_LOOPS_NUM; i++) {
		CZ_CUDA_CALL(hipMemcpy(memDevice2, memDevice1, CZ_BAND_BUF_SIZE, hipMemcpyDeviceToDevice),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
	}

	CZ_CUDA_CALL(hipEventRecord(stop, 0),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventSynchronize(stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	CZ_CUDA_CALL(hipEventElapsedTime(&timeMs, start, stop),
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return 0);

	printf("Test complete in %f ms.\n", timeMs);

	bandwidthKBs = (1000 * (float)CZ_BAND_BUF_SIZE * (float)CZ_BAND_LOOPS_NUM) / (timeMs * (float)(1 << 10));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)bandwidthKBs;
}

/*!
	\brief Run several bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidthTest(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	info->band.copyHDPage = cudaCalcDeviceBandwidthTestHD(info, 0);
	info->band.copyHDPin = cudaCalcDeviceBandwidthTestHD(info, 1);
	info->band.copyDHPage = cudaCalcDeviceBandwidthTestDH(info, 0);
	info->band.copyDHPin = cudaCalcDeviceBandwidthTestDH(info, 1);
	info->band.copyDD = cudaCalcDeviceBandwidthTestDD(info);

	return 0;
}

/*!
	\brief Prepare buffers bandwidth tests.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaPrepareDevice(
	struct CZDeviceInfo *info
) {

	if(info == NULL)
		return -1;

	if(!cudaIsInit())
		return -1;

	if(cudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	return 0;
}

/*!
	\brief Calculate bandwidth information about CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCalcDeviceBandwidth(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

//	printf("cudaCalcDeviceBandwidth called!\n");

	if(info == NULL)
		return -1;

	if(cudaCalcDeviceBandwidthReset(info) != 0)
		return -1;

	if(!cudaIsInit())
		return -1;

	if(cudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	if(cudaCalcDeviceBandwidthTest(info) != 0)
		return -1;

	return 0;
}

/*!
	\brief Cleanup after test and bandwidth calculations.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int cudaCleanDevice(
	struct CZDeviceInfo *info	/*!< CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(cudaCalcDeviceBandwidthFree(info) != 0)
		return -1;

	return 0;
}
